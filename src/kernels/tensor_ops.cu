#include "hip/hip_runtime.h"
// tensor_ops.cu
// Operation functors and launch helper functions for tensor operations

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <stdio.h>
#include <algorithm>
#include <vector>
#include "kernel_utils.cuh"
#include "tensor_kernels.cuh"
#include "../cuda_utils.h"
#include <hipblas.h>

// Operation functors are now defined in tensor_kernels.cuh

// Helper macros for common kernel launch patterns
#define LAUNCH_BINARY_KERNEL(result, a, b, n, op) \
    do { \
        int threadsPerBlock = 256; \
        int blocksPerGrid = ((n) + threadsPerBlock - 1) / threadsPerBlock; \
        elementwise_binary_kernel<<<blocksPerGrid, threadsPerBlock>>>((result), (a), (b), (n), (op)); \
        hipDeviceSynchronize(); \
    } while(0)

#define LAUNCH_UNARY_KERNEL(result, input, n, op) \
    do { \
        int threadsPerBlock = 256; \
        int blocksPerGrid = ((n) + threadsPerBlock - 1) / threadsPerBlock; \
        elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>((result), (input), (n), (op)); \
        hipDeviceSynchronize(); \
    } while(0)

#define LAUNCH_SCALAR_KERNEL(result, input, scalar, n, op) \
    do { \
        int threadsPerBlock = 256; \
        int blocksPerGrid = ((n) + threadsPerBlock - 1) / threadsPerBlock; \
        elementwise_scalar_kernel<<<blocksPerGrid, threadsPerBlock>>>((result), (input), (scalar), (n), (op)); \
        hipDeviceSynchronize(); \
    } while(0)

template<typename To, typename From>
void launch_type_conversion(To* output, const From* input, size_t n, hipStream_t stream = 0) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    
    type_conversion_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(output, input, n);
}

template<typename T, typename AccumType, typename Op>
AccumType launch_reduction(const T* input, size_t n, Op op, AccumType init_val, hipStream_t stream = 0) {
    // Handle edge cases
    if (n == 0) return init_val;
    if (n == 1) {
        // For single element, copy from device to host safely
        T single_element;
        hipError_t cudaStatus = hipMemcpy(&single_element, input, sizeof(T), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            printf("CUDA memcpy failed for single element: %s\n", hipGetErrorString(cudaStatus));
            return init_val;
        }
        return convert_type<AccumType>(single_element);
    }
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    
    // Limit the number of blocks to prevent excessive memory usage
    blocksPerGrid = std::min(blocksPerGrid, 65535);
    
    AccumType* d_block_results = nullptr;
    hipError_t cudaStatus = hipMalloc(&d_block_results, blocksPerGrid * sizeof(AccumType));
    if (cudaStatus != hipSuccess) {
        printf("CUDA malloc failed: %s\n", hipGetErrorString(cudaStatus));
        return init_val;
    }
    
    size_t sharedMemSize = threadsPerBlock * sizeof(AccumType);
    reduction_kernel<<<blocksPerGrid, threadsPerBlock, sharedMemSize, stream>>>(
        d_block_results, input, n, op, init_val
    );
    
    // Check for kernel launch errors
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        printf("CUDA kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_block_results);
        return init_val;
    }
    
    // Wait for kernel completion
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        printf("CUDA synchronize failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_block_results);
        return init_val;
    }
    
    // Copy results back and reduce on host
    AccumType* h_block_results = (AccumType*)malloc(blocksPerGrid * sizeof(AccumType));
    if (!h_block_results) {
        printf("Host malloc failed\n");
        hipFree(d_block_results);
        return init_val;
    }
    
    cudaStatus = hipMemcpy(h_block_results, d_block_results, blocksPerGrid * sizeof(AccumType), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        printf("CUDA memcpy failed: %s\n", hipGetErrorString(cudaStatus));
        free(h_block_results);
        hipFree(d_block_results);
        return init_val;
    }
    
    // Host-side final reduction - now this will work with __host__ __device__ functors!
    AccumType final_result = h_block_results[0];
    for (int i = 1; i < blocksPerGrid; i++) {
        final_result = op(final_result, h_block_results[i]);
    }
    
    // Cleanup
    free(h_block_results);
    hipFree(d_block_results);
    
    return final_result;
}

template<typename T>
void launch_matmul(T* C, const T* A, const T* B, size_t M, size_t N, size_t K, hipStream_t stream = 0) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    // Use tiled version for better performance
    matmul_tiled_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(C, A, B, M, N, K);
}

// Launch outer product: result[i,j] = a[i] * b[j]
template<typename T>
void launch_outer_product(T* result, const T* a, const T* b, size_t M, size_t N, hipStream_t stream = 0) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    outer_product_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(result, a, b, M, N);
}

// Launch matrix-vector multiplication: result[i] = sum_j(A[i,j] * v[j])
template<typename T>
void launch_matvec(T* result, const T* A, const T* v, size_t M, size_t N, hipStream_t stream = 0) {
    int blockSize = 256;
    int gridSize = (M + blockSize - 1) / blockSize;
    
    // Temporarily use simple version for debugging
    matvec_kernel<<<gridSize, blockSize, 0, stream>>>(result, A, v, M, N);
}

// Launch vector-matrix multiplication: result[j] = sum_i(v[i] * A[i,j])
template<typename T>
void launch_vecmat(T* result, const T* v, const T* A, size_t M, size_t N, hipStream_t stream = 0) {
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;
    
    vecmat_kernel<<<gridSize, blockSize, 0, stream>>>(result, v, A, M, N);
}

// Launch helper for broadcast operations
template<typename T, typename Op>
void launch_broadcast_binary(
    T* result, const T* a, const T* b,
    const int* a_strides, const int* b_strides, const int* result_strides,
    const int* shape, int ndims, size_t total_elements, Op op
) {
    // Copy stride and shape data to device
    int *d_a_strides, *d_b_strides, *d_result_strides, *d_shape;
    hipMalloc(&d_a_strides, ndims * sizeof(int));
    hipMalloc(&d_b_strides, ndims * sizeof(int));
    hipMalloc(&d_result_strides, ndims * sizeof(int));
    hipMalloc(&d_shape, ndims * sizeof(int));
    
    hipMemcpy(d_a_strides, a_strides, ndims * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, ndims * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_result_strides, result_strides, ndims * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_shape, shape, ndims * sizeof(int), hipMemcpyHostToDevice);
    
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    broadcast_binary_kernel<T, Op><<<grid_size, block_size>>>(
        result, a, b, d_a_strides, d_b_strides, d_result_strides, d_shape, ndims, total_elements
    );
    
    hipFree(d_a_strides);
    hipFree(d_b_strides);
    hipFree(d_result_strides);
    hipFree(d_shape);
}

template<typename T>
void launch_strided_copy(T* dest, const T* src, const std::vector<int>& strides, const std::vector<int>& shape, size_t total_elements) {
    // Copy stride and shape data to device
    int* d_strides;
    int* d_shape;
    int ndims = shape.size();
    
    hipMalloc(&d_strides, ndims * sizeof(int));
    hipMalloc(&d_shape, ndims * sizeof(int));
    
    hipMemcpy(d_strides, strides.data(), ndims * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_shape, shape.data(), ndims * sizeof(int), hipMemcpyHostToDevice);
    
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_copy_kernel<T><<<grid_size, block_size>>>(
        dest, src, d_strides, d_shape, ndims, total_elements
    );
    
    hipFree(d_strides);
    hipFree(d_shape);
} 

// Launch concat operation
template<typename T>
void launch_concat(T* result, const T** inputs, const int* input_sizes, int num_tensors,
                   const std::vector<int>& result_strides, const std::vector<std::vector<int>>& input_strides_list,
                   const std::vector<int>& shape, int concat_axis, size_t total_elements) {
    
    // Prepare device memory for strides
    int* d_result_strides;
    int* d_input_strides_list;
    int* d_input_sizes;
    int* d_shape;
    T** d_inputs;
    
    int ndims = shape.size();
    
    hipMalloc(&d_result_strides, ndims * sizeof(int));
    hipMalloc(&d_input_strides_list, num_tensors * ndims * sizeof(int));
    hipMalloc(&d_input_sizes, num_tensors * sizeof(int));
    hipMalloc(&d_shape, ndims * sizeof(int));
    hipMalloc(&d_inputs, num_tensors * sizeof(T*));
    
    hipMemcpy(d_result_strides, result_strides.data(), ndims * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_input_sizes, input_sizes, num_tensors * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_shape, shape.data(), ndims * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_inputs, inputs, num_tensors * sizeof(T*), hipMemcpyHostToDevice);
    
    // Flatten input strides
    std::vector<int> flattened_strides;
    for (const auto& strides : input_strides_list) {
        flattened_strides.insert(flattened_strides.end(), strides.begin(), strides.end());
    }
    hipMemcpy(d_input_strides_list, flattened_strides.data(), num_tensors * ndims * sizeof(int), hipMemcpyHostToDevice);
    
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    concat_kernel<T><<<grid_size, block_size>>>(
        result, d_inputs, d_input_sizes, num_tensors,
        d_result_strides, d_input_strides_list, d_shape, ndims, concat_axis, total_elements
    );
    
    hipFree(d_result_strides);
    hipFree(d_input_strides_list);
    hipFree(d_input_sizes);
    hipFree(d_shape);
    hipFree(d_inputs);
}

// Launch stack operation
template<typename T>
void launch_stack(T* result, const T** inputs, int num_tensors,
                  const std::vector<int>& input_strides, const std::vector<int>& result_shape,
                  int stack_axis, size_t total_elements) {
    
    int* d_input_strides;
    int* d_result_shape;
    T** d_inputs;
    
    int ndims = result_shape.size();
    
    hipMalloc(&d_input_strides, (ndims-1) * sizeof(int));
    hipMalloc(&d_result_shape, ndims * sizeof(int));
    hipMalloc(&d_inputs, num_tensors * sizeof(T*));
    
    hipMemcpy(d_input_strides, input_strides.data(), (ndims-1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_result_shape, result_shape.data(), ndims * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_inputs, inputs, num_tensors * sizeof(T*), hipMemcpyHostToDevice);
    
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    stack_kernel<T><<<grid_size, block_size>>>(
        result, d_inputs, num_tensors, d_input_strides, d_result_shape, ndims, stack_axis, total_elements
    );
    
    hipFree(d_input_strides);
    hipFree(d_result_shape);
    hipFree(d_inputs);
}

// Launch repeat operation
template<typename T>
void launch_repeat(T* result, const T* input,
                   const std::vector<int>& input_strides, const std::vector<int>& repeat_counts,
                   const std::vector<int>& input_shape, const std::vector<int>& result_shape,
                   size_t total_elements) {
    
    int* d_input_strides;
    int* d_repeat_counts;
    int* d_input_shape;
    int* d_result_shape;
    
    int ndims = input_shape.size();
    
    hipMalloc(&d_input_strides, ndims * sizeof(int));
    hipMalloc(&d_repeat_counts, ndims * sizeof(int));
    hipMalloc(&d_input_shape, ndims * sizeof(int));
    hipMalloc(&d_result_shape, ndims * sizeof(int));
    
    hipMemcpy(d_input_strides, input_strides.data(), ndims * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_repeat_counts, repeat_counts.data(), ndims * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_input_shape, input_shape.data(), ndims * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_result_shape, result_shape.data(), ndims * sizeof(int), hipMemcpyHostToDevice);
    
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    repeat_kernel<T><<<grid_size, block_size>>>(
        result, input, d_input_strides, d_repeat_counts, d_input_shape, d_result_shape, ndims, total_elements
    );
    
    hipFree(d_input_strides);
    hipFree(d_repeat_counts);
    hipFree(d_input_shape);
    hipFree(d_result_shape);
}

// Launch pad operation
template<typename T>
void launch_pad(T* result, const T* input,
                const std::vector<int>& input_strides, const std::vector<int>& input_shape,
                const std::vector<int>& pad_before, const std::vector<int>& pad_after,
                const std::vector<int>& result_shape, T pad_value, int pad_mode, size_t total_elements) {
    
    int* d_input_strides;
    int* d_input_shape;
    int* d_pad_before;
    int* d_pad_after;
    int* d_result_shape;
    
    int ndims = input_shape.size();
    
    hipMalloc(&d_input_strides, ndims * sizeof(int));
    hipMalloc(&d_input_shape, ndims * sizeof(int));
    hipMalloc(&d_pad_before, ndims * sizeof(int));
    hipMalloc(&d_pad_after, ndims * sizeof(int));
    hipMalloc(&d_result_shape, ndims * sizeof(int));
    
    hipMemcpy(d_input_strides, input_strides.data(), ndims * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_input_shape, input_shape.data(), ndims * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_pad_before, pad_before.data(), ndims * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_pad_after, pad_after.data(), ndims * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_result_shape, result_shape.data(), ndims * sizeof(int), hipMemcpyHostToDevice);
    
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    pad_kernel<T><<<grid_size, block_size>>>(
        result, input, d_input_strides, d_input_shape, d_pad_before, d_pad_after, 
        d_result_shape, ndims, pad_value, pad_mode, total_elements
    );
    
    hipFree(d_input_strides);
    hipFree(d_input_shape);
    hipFree(d_pad_before);
    hipFree(d_pad_after);
    hipFree(d_result_shape);
} 

// ===================== Product and Variance Support ===================== //
// SquareOp is now defined in tensor_kernels.cuh

// ===================== Softmax / Argmax Kernels ===================== //

template<typename T>
__global__ void shift_exp_kernel(T* out, const T* in, T shift, size_t n) {
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        out[tid] = exp(in[tid] - shift);
    }
}

template<typename T>
__global__ void div_scalar_kernel(T* data, T scalar, size_t n) {
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        data[tid] /= scalar;
    }
}

// ===================== Strided Operation Kernels ===================== //

// Strided kernels for non-contiguous tensor operations
template<typename T, typename Op>
__global__ void strided_unary_kernel(
    cuda_utils::TensorDescriptor out_desc,
    cuda_utils::TensorDescriptor in_desc,
    size_t total_elements
) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total_elements) return;
    
    // Convert linear index to coordinates
    int coords[8];
    out_desc.linear_to_coords(idx, coords);
    
    // Compute offsets using strides
    size_t in_offset = in_desc.compute_offset(coords);
    size_t out_offset = out_desc.compute_offset(coords);
    
    // Perform operation
    T* out_ptr = static_cast<T*>(out_desc.data);
    const T* in_ptr = static_cast<const T*>(in_desc.data);
    
    out_ptr[out_offset] = Op{}(in_ptr[in_offset]);
}

template<typename T, typename Op>
__global__ void strided_binary_kernel(
    cuda_utils::TensorDescriptor out_desc,
    cuda_utils::TensorDescriptor a_desc,
    cuda_utils::TensorDescriptor b_desc,
    size_t total_elements
) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total_elements) return;
    
    // Convert linear index to coordinates
    int coords[8];
    out_desc.linear_to_coords(idx, coords);
    
    // Compute offsets using strides
    size_t a_offset = a_desc.compute_offset(coords);
    size_t b_offset = b_desc.compute_offset(coords);
    size_t out_offset = out_desc.compute_offset(coords);
    
    // Perform operation
    T* out_ptr = static_cast<T*>(out_desc.data);
    const T* a_ptr = static_cast<const T*>(a_desc.data);
    const T* b_ptr = static_cast<const T*>(b_desc.data);
    
    out_ptr[out_offset] = Op{}(a_ptr[a_offset], b_ptr[b_offset]);
}

// Strided scalar operation kernel
template<typename T, typename U, typename Op>
__global__ void strided_scalar_kernel(
    cuda_utils::TensorDescriptor out_desc,
    cuda_utils::TensorDescriptor in_desc,
    U scalar,
    size_t total_elements
) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total_elements) return;
    
    // Convert linear index to coordinates
    int coords[8];
    out_desc.linear_to_coords(idx, coords);
    
    // Compute offsets using strides
    size_t in_offset = in_desc.compute_offset(coords);
    size_t out_offset = out_desc.compute_offset(coords);
    
    // Perform operation
    T* out_ptr = static_cast<T*>(out_desc.data);
    const T* in_ptr = static_cast<const T*>(in_desc.data);
    
    out_ptr[out_offset] = Op{}(in_ptr[in_offset], static_cast<T>(scalar));
}

// BEGIN NEW WRAPPER FUNCTIONS (Sprint 1 enhancements)
extern "C" {

// ===================== Unary Elementwise Math ===================== //

void tensor_exp_float32(float* result, const float* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, ExpOp());
    hipDeviceSynchronize();
}

void tensor_exp_float64(double* result, const double* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, ExpOp());
    hipDeviceSynchronize();
}

void tensor_log_float32(float* result, const float* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, LogOp());
    hipDeviceSynchronize();
}

void tensor_log_float64(double* result, const double* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, LogOp());
    hipDeviceSynchronize();
}

void tensor_sqrt_float32(float* result, const float* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, SqrtOp());
    hipDeviceSynchronize();
}

void tensor_sqrt_float64(double* result, const double* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, SqrtOp());
    hipDeviceSynchronize();
}

// New activation functions
void tensor_tanh_float32(float* result, const float* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, TanhOp());
    hipDeviceSynchronize();
}

void tensor_tanh_float64(double* result, const double* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, TanhOp());
    hipDeviceSynchronize();
}

void tensor_sigmoid_float32(float* result, const float* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, SigmoidOp());
    hipDeviceSynchronize();
}

void tensor_sigmoid_float64(double* result, const double* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, SigmoidOp());
    hipDeviceSynchronize();
}

void tensor_relu_float32(float* result, const float* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, ReluOp());
    hipDeviceSynchronize();
}

void tensor_relu_float64(double* result, const double* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, ReluOp());
    hipDeviceSynchronize();
}

void tensor_sin_float32(float* result, const float* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, SinOp());
    hipDeviceSynchronize();
}

void tensor_sin_float64(double* result, const double* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, SinOp());
    hipDeviceSynchronize();
}

void tensor_cos_float32(float* result, const float* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, CosOp());
    hipDeviceSynchronize();
}

void tensor_cos_float64(double* result, const double* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, CosOp());
    hipDeviceSynchronize();
}

// New math functions for Phase 3.1
void tensor_floor_float32(float* result, const float* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, FloorOp());
    hipDeviceSynchronize();
}

void tensor_floor_float64(double* result, const double* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, FloorOp());
    hipDeviceSynchronize();
}

void tensor_ceil_float32(float* result, const float* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, CeilOp());
    hipDeviceSynchronize();
}

void tensor_ceil_float64(double* result, const double* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, CeilOp());
    hipDeviceSynchronize();
}

void tensor_round_float32(float* result, const float* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, RoundOp());
    hipDeviceSynchronize();
}

void tensor_round_float64(double* result, const double* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, RoundOp());
    hipDeviceSynchronize();
}

void tensor_erf_float32(float* result, const float* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, ErfOp());
    hipDeviceSynchronize();
}

void tensor_erf_float64(double* result, const double* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, ErfOp());
    hipDeviceSynchronize();
}

// Scalar power operation
void tensor_pow_scalar_float32(float* result, const float* input, float exponent, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_scalar_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, exponent, n, PowScalarOp());
    hipDeviceSynchronize();
}

void tensor_pow_scalar_float64(double* result, const double* input, double exponent, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_scalar_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, exponent, n, PowScalarOp());
    hipDeviceSynchronize();
}

void tensor_abs_float32(float* result, const float* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, AbsOp());
    hipDeviceSynchronize();
}

void tensor_abs_float64(double* result, const double* input, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, n, AbsOp());
    hipDeviceSynchronize();
}

// ===================== Reductions ===================== //

#include <float.h>
#include <cfloat>

// Sum (FLOAT32 & FLOAT64) - using AddOp from above
float tensor_sum_float32(const float* input, size_t n) {
    return launch_reduction<float, float>(input, n, AddOp(), 0.0f);
}

double tensor_sum_float64(const double* input, size_t n) {
    return launch_reduction<double, double>(input, n, AddOp(), 0.0);
}

// Placeholder for half precision & int64 sums (not yet optimized)
float tensor_sum_float16(const half* /*input*/, size_t /*n*/) {
    // TODO: Implement half precision reduction
    return 0.0f;
}

int64_t tensor_sum_int64(const int64_t* /*input*/, size_t /*n*/) {
    // TODO: Implement int64 reduction
    return 0;
}

// Max
float tensor_max_float32(const float* input, size_t n) {
    return launch_reduction<float, float>(input, n, MaxOp(), -FLT_MAX);
}

double tensor_max_float64(const double* input, size_t n) {
    return launch_reduction<double, double>(input, n, MaxOp(), -DBL_MAX);
}

// Min
float tensor_min_float32(const float* input, size_t n) {
    return launch_reduction<float, float>(input, n, MinOp(), FLT_MAX);
}

double tensor_min_float64(const double* input, size_t n) {
    return launch_reduction<double, double>(input, n, MinOp(), DBL_MAX);
}

// ===================== Comparison Functors ===================== //

void tensor_gt_float32(float* result, const float* a, const float* b, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_binary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, a, b, n, GreaterOp());
    hipDeviceSynchronize();
}

void tensor_gt_float64(double* result, const double* a, const double* b, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_binary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, a, b, n, GreaterOp());
    hipDeviceSynchronize();
}

void tensor_lt_float32(float* result, const float* a, const float* b, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_binary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, a, b, n, LessOp());
    hipDeviceSynchronize();
}

void tensor_lt_float64(double* result, const double* a, const double* b, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_binary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, a, b, n, LessOp());
    hipDeviceSynchronize();
}

void tensor_eq_float32(float* result, const float* a, const float* b, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_binary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, a, b, n, EqualOp());
    hipDeviceSynchronize();
}

void tensor_eq_float64(double* result, const double* a, const double* b, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_binary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, a, b, n, EqualOp());
    hipDeviceSynchronize();
}

// Product reductions
float tensor_prod_float32(const float* input, size_t n) {
    return launch_reduction<float, float>(input, n, MulOp(), 1.0f);
}

double tensor_prod_float64(const double* input, size_t n) {
    return launch_reduction<double, double>(input, n, MulOp(), 1.0);
}

// Variance (population) - returns double for precision

double tensor_var_float32(const float* input, size_t n) {
    if (n <= 1) return 0.0;          // Variance undefined for n <= 1, return 0
    // Allocate temp buffer for squares
    float* d_squares = nullptr;
    hipMalloc(&d_squares, n * sizeof(float));
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_squares, input, n, SquareOp());
    hipDeviceSynchronize();
    
    float sum = launch_reduction<float, float>(input, n, AddOp(), 0.0f);
    float sum_sq = launch_reduction<float, float>(d_squares, n, AddOp(), 0.0f);
    hipFree(d_squares);
    double mean = static_cast<double>(sum) / static_cast<double>(n);
    // Sample variance with Bessel's correction
    double numerator = static_cast<double>(sum_sq) - static_cast<double>(n) * mean * mean;
    return numerator / static_cast<double>(n - 1);
}

double tensor_var_float64(const double* input, size_t n) {
    if (n <= 1) return 0.0;
    // Allocate temp buffer for squares
    double* d_squares = nullptr;
    hipMalloc(&d_squares, n * sizeof(double));
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_unary_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_squares, input, n, SquareOp());
    hipDeviceSynchronize();
    
    double sum = launch_reduction<double, double>(input, n, AddOp(), 0.0);
    double sum_sq = launch_reduction<double, double>(d_squares, n, AddOp(), 0.0);
    hipFree(d_squares);
    double mean = sum / static_cast<double>(n);
    double numerator = sum_sq - static_cast<double>(n) * mean * mean;
    return numerator / static_cast<double>(n - 1);
}

// ----------------- Softmax wrappers -----------------

void tensor_softmax_float32(float* output, const float* input, size_t n) {
    // 1. max
    float max_val = launch_reduction<float,float>(input, n, MaxOp(), -FLT_MAX);
    // 2. exp(x-max)
    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    shift_exp_kernel<float><<<blocks, threads>>>(output, input, max_val, n);
    // 3. sum
    float sum_val = launch_reduction<float,float>(output, n, AddOp(), 0.0f);
    // 4. divide
    div_scalar_kernel<float><<<blocks, threads>>>(output, sum_val, n);
}

void tensor_softmax_float64(double* output, const double* input, size_t n) {
    double max_val = launch_reduction<double,double>(input, n, MaxOp(), -DBL_MAX);
    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    shift_exp_kernel<double><<<blocks, threads>>>(output, input, max_val, n);
    double sum_val = launch_reduction<double,double>(output, n, AddOp(), 0.0);
    div_scalar_kernel<double><<<blocks, threads>>>(output, sum_val, n);
}

// ----------------- Argmax wrappers -----------------

int64_t tensor_argmax_float32(const float* input, size_t n) {
    std::vector<float> host(n);
    hipMemcpy(host.data(), input, n*sizeof(float), hipMemcpyDeviceToHost);
    auto it = std::max_element(host.begin(), host.end());
    return static_cast<int64_t>(std::distance(host.begin(), it));
}

int64_t tensor_argmax_float64(const double* input, size_t n) {
    std::vector<double> host(n);
    hipMemcpy(host.data(), input, n*sizeof(double), hipMemcpyDeviceToHost);
    auto it = std::max_element(host.begin(), host.end());
    return static_cast<int64_t>(std::distance(host.begin(), it));
}

// ===================== C Wrappers for TensorMutation.cpp ===================== //

void launch_concat_float32(float* result, const float** inputs, const int* input_sizes, int num_tensors,
                           const int* result_strides, const int* input_strides_list, 
                           const int* shape, int ndims, int concat_axis, size_t total_elements) {
    std::vector<int> result_strides_vec(result_strides, result_strides + ndims);
    std::vector<std::vector<int>> input_strides_list_vec(num_tensors);
    for (int i = 0; i < num_tensors; ++i) {
        input_strides_list_vec[i] = std::vector<int>(input_strides_list + i * ndims, 
                                                     input_strides_list + (i + 1) * ndims);
    }
    std::vector<int> shape_vec(shape, shape + ndims);
    launch_concat<float>(result, inputs, input_sizes, num_tensors,
                         result_strides_vec, input_strides_list_vec, shape_vec, concat_axis, total_elements);
}

void launch_concat_float64(double* result, const double** inputs, const int* input_sizes, int num_tensors,
                           const int* result_strides, const int* input_strides_list, 
                           const int* shape, int ndims, int concat_axis, size_t total_elements) {
    std::vector<int> result_strides_vec(result_strides, result_strides + ndims);
    std::vector<std::vector<int>> input_strides_list_vec(num_tensors);
    for (int i = 0; i < num_tensors; ++i) {
        input_strides_list_vec[i] = std::vector<int>(input_strides_list + i * ndims, 
                                                     input_strides_list + (i + 1) * ndims);
    }
    std::vector<int> shape_vec(shape, shape + ndims);
    launch_concat<double>(result, inputs, input_sizes, num_tensors,
                          result_strides_vec, input_strides_list_vec, shape_vec, concat_axis, total_elements);
}

void launch_stack_float32(float* result, const float** inputs, int num_tensors,
                          const int* input_strides, const int* result_shape, int ndims,
                          int stack_axis, size_t total_elements) {
    std::vector<int> input_strides_vec(input_strides, input_strides + (ndims - 1));
    std::vector<int> result_shape_vec(result_shape, result_shape + ndims);
    launch_stack<float>(result, inputs, num_tensors, input_strides_vec, result_shape_vec, stack_axis, total_elements);
}

void launch_stack_float64(double* result, const double** inputs, int num_tensors,
                          const int* input_strides, const int* result_shape, int ndims,
                          int stack_axis, size_t total_elements) {
    std::vector<int> input_strides_vec(input_strides, input_strides + (ndims - 1));
    std::vector<int> result_shape_vec(result_shape, result_shape + ndims);
    launch_stack<double>(result, inputs, num_tensors, input_strides_vec, result_shape_vec, stack_axis, total_elements);
}

void launch_repeat_float32(float* result, const float* input,
                           const int* input_strides, const int* repeat_counts,
                           const int* input_shape, const int* result_shape, int ndims,
                           size_t total_elements) {
    std::vector<int> input_strides_vec(input_strides, input_strides + ndims);
    std::vector<int> repeat_counts_vec(repeat_counts, repeat_counts + ndims);
    std::vector<int> input_shape_vec(input_shape, input_shape + ndims);
    std::vector<int> result_shape_vec(result_shape, result_shape + ndims);
    launch_repeat<float>(result, input, input_strides_vec, repeat_counts_vec, input_shape_vec, result_shape_vec, total_elements);
}

void launch_repeat_float64(double* result, const double* input,
                           const int* input_strides, const int* repeat_counts,
                           const int* input_shape, const int* result_shape, int ndims,
                           size_t total_elements) {
    std::vector<int> input_strides_vec(input_strides, input_strides + ndims);
    std::vector<int> repeat_counts_vec(repeat_counts, repeat_counts + ndims);
    std::vector<int> input_shape_vec(input_shape, input_shape + ndims);
    std::vector<int> result_shape_vec(result_shape, result_shape + ndims);
    launch_repeat<double>(result, input, input_strides_vec, repeat_counts_vec, input_shape_vec, result_shape_vec, total_elements);
}

void launch_pad_float32(float* result, const float* input,
                        const int* input_strides, const int* input_shape,
                        const int* pad_before, const int* pad_after,
                        const int* result_shape, int ndims, float pad_value, int pad_mode,
                        size_t total_elements) {
    std::vector<int> input_strides_vec(input_strides, input_strides + ndims);
    std::vector<int> input_shape_vec(input_shape, input_shape + ndims);
    std::vector<int> pad_before_vec(pad_before, pad_before + ndims);
    std::vector<int> pad_after_vec(pad_after, pad_after + ndims);
    std::vector<int> result_shape_vec(result_shape, result_shape + ndims);
    launch_pad<float>(result, input, input_strides_vec, input_shape_vec, pad_before_vec, pad_after_vec, 
                      result_shape_vec, pad_value, pad_mode, total_elements);
}

void launch_pad_float64(double* result, const double* input,
                        const int* input_strides, const int* input_shape,
                        const int* pad_before, const int* pad_after,
                        const int* result_shape, int ndims, double pad_value, int pad_mode,
                        size_t total_elements) {
    std::vector<int> input_strides_vec(input_strides, input_strides + ndims);
    std::vector<int> input_shape_vec(input_shape, input_shape + ndims);
    std::vector<int> pad_before_vec(pad_before, pad_before + ndims);
    std::vector<int> pad_after_vec(pad_after, pad_after + ndims);
    std::vector<int> result_shape_vec(result_shape, result_shape + ndims);
    launch_pad<double>(result, input, input_strides_vec, input_shape_vec, pad_before_vec, pad_after_vec, 
                       result_shape_vec, pad_value, pad_mode, total_elements);
}

void tensor_div_float16(half* result, const half* a, const half* b, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_binary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, a, b, n, DivOp());
    hipDeviceSynchronize();
}

// ===================== Missing Arithmetic Functions ===================== //

// Multiplication operations
void tensor_mul_float16(half* result, const half* a, const half* b, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_binary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, a, b, n, MulOp());
    hipDeviceSynchronize();
}

void tensor_mul_float32(float* result, const float* a, const float* b, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_binary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, a, b, n, MulOp());
    hipDeviceSynchronize();
}

void tensor_mul_float64(double* result, const double* a, const double* b, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_binary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, a, b, n, MulOp());
    hipDeviceSynchronize();
}

// Subtraction operations
void tensor_sub_float16(half* result, const half* a, const half* b, size_t n) {
    LAUNCH_BINARY_KERNEL(result, a, b, n, SubOp());
}

void tensor_sub_float32(float* result, const float* a, const float* b, size_t n) {
    LAUNCH_BINARY_KERNEL(result, a, b, n, SubOp());
}

void tensor_sub_float64(double* result, const double* a, const double* b, size_t n) {
    LAUNCH_BINARY_KERNEL(result, a, b, n, SubOp());
}

// Division operations
void tensor_div_float32(float* result, const float* a, const float* b, size_t n) {
    LAUNCH_BINARY_KERNEL(result, a, b, n, DivOp());
}

void tensor_div_float64(double* result, const double* a, const double* b, size_t n) {
    LAUNCH_BINARY_KERNEL(result, a, b, n, DivOp());
}

// Scalar operations
void tensor_scalar_mul_float16(half* result, const half* input, float scalar, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_scalar_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, scalar, n, MulOp());
    hipDeviceSynchronize();
}

void tensor_scalar_mul_float32(float* result, const float* input, float scalar, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_scalar_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, scalar, n, MulOp());
    hipDeviceSynchronize();
}

void tensor_scalar_mul_float64(double* result, const double* input, double scalar, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_scalar_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, scalar, n, MulOp());
    hipDeviceSynchronize();
}

void tensor_scalar_add_float16(half* result, const half* input, float scalar, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_scalar_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, scalar, n, AddOp());
    hipDeviceSynchronize();
}

void tensor_scalar_add_float32(float* result, const float* input, float scalar, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_scalar_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, scalar, n, AddOp());
    hipDeviceSynchronize();
}

void tensor_scalar_add_float64(double* result, const double* input, double scalar, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_scalar_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, scalar, n, AddOp());
    hipDeviceSynchronize();
}

// ===================== Additional Missing Functions ===================== //

// Addition operations
void tensor_add_float16(half* result, const half* a, const half* b, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_binary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, a, b, n, AddOp());
    hipDeviceSynchronize();
}

void tensor_add_float32(float* result, const float* a, const float* b, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_binary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, a, b, n, AddOp());
    hipDeviceSynchronize();
}

void tensor_add_float64(double* result, const double* a, const double* b, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_binary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, a, b, n, AddOp());
    hipDeviceSynchronize();
}

void tensor_add_int8(int8_t* result, const int8_t* a, const int8_t* b, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_binary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, a, b, n, AddOp());
    hipDeviceSynchronize();
}

void tensor_add_int32(int32_t* result, const int32_t* a, const int32_t* b, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_binary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, a, b, n, AddOp());
    hipDeviceSynchronize();
}

void tensor_add_int64(int64_t* result, const int64_t* a, const int64_t* b, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_binary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, a, b, n, AddOp());
    hipDeviceSynchronize();
}

// Broadcast operations
void tensor_add_broadcast_float32(float* result, const float* a, const float* b, const int* a_strides, 
                                  const int* b_strides, const int* result_strides, const int* shape, 
                                  int ndims, size_t total_elements) {
    launch_broadcast_binary<float>(result, a, b, a_strides, b_strides, result_strides, shape, ndims, total_elements, AddOp());
}

void tensor_add_broadcast_float64(double* result, const double* a, const double* b, const int* a_strides, 
                                  const int* b_strides, const int* result_strides, const int* shape, 
                                  int ndims, size_t total_elements) {
    launch_broadcast_binary<double>(result, a, b, a_strides, b_strides, result_strides, shape, ndims, total_elements, AddOp());
}

void tensor_mul_broadcast_float32(float* result, const float* a, const float* b, const int* a_strides, 
                                  const int* b_strides, const int* result_strides, const int* shape, 
                                  int ndims, size_t total_elements) {
    launch_broadcast_binary<float>(result, a, b, a_strides, b_strides, result_strides, shape, ndims, total_elements, MulOp());
}

void tensor_mul_broadcast_float64(double* result, const double* a, const double* b, const int* a_strides, 
                                  const int* b_strides, const int* result_strides, const int* shape, 
                                  int ndims, size_t total_elements) {
    launch_broadcast_binary<double>(result, a, b, a_strides, b_strides, result_strides, shape, ndims, total_elements, MulOp());
}

// Matrix multiplication operations
void tensor_matmul_float16(half* C, const half* A, const half* B, size_t M, size_t N, size_t K) {
    const float alpha = 1.0f;
    const float beta  = 0.0f;
    hipblasHandle_t handle = cuda_utils::get_cublas_handle();
    hipblasStatus_t stat = hipblasGemmEx(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        static_cast<int>(M), static_cast<int>(N), static_cast<int>(K),
        &alpha,
        A, HIP_R_16F, static_cast<int>(M),
        B, HIP_R_16F, static_cast<int>(K),
        &beta,
        C, HIP_R_16F, static_cast<int>(M),
        HIP_R_32F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        throw std::runtime_error("hipblasGemmEx failed in tensor_matmul_float16");
    }
}

void tensor_matmul_float32(float* C, const float* A, const float* B, size_t M, size_t N, size_t K) {
    const float alpha = 1.0f;
    const float beta  = 0.0f;
    hipblasHandle_t handle = cuda_utils::get_cublas_handle();
    hipblasStatus_t stat = hipblasGemmEx(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        static_cast<int>(M), static_cast<int>(N), static_cast<int>(K),
        &alpha,
        A, HIP_R_32F, static_cast<int>(M),
        B, HIP_R_32F, static_cast<int>(K),
        &beta,
        C, HIP_R_32F, static_cast<int>(M),
        HIP_R_32F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        throw std::runtime_error("hipblasGemmEx failed in tensor_matmul_float32");
    }
}

void tensor_matmul_float64(double* C, const double* A, const double* B, size_t M, size_t N, size_t K) {
    const double alpha = 1.0;
    const double beta  = 0.0;
    hipblasHandle_t handle = cuda_utils::get_cublas_handle();
    hipblasStatus_t stat = hipblasDgemm(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        static_cast<int>(M), static_cast<int>(N), static_cast<int>(K),
        &alpha,
        A, static_cast<int>(M),
        B, static_cast<int>(K),
        &beta,
        C, static_cast<int>(M));
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        throw std::runtime_error("hipblasDgemm failed in tensor_matmul_float64");
    }
}

// Outer product operations
void tensor_outer_product_float16(half* result, const half* a, const half* b, size_t M, size_t N) {
    launch_outer_product<half>(result, a, b, M, N);
}

void tensor_outer_product_float32(float* result, const float* a, const float* b, size_t M, size_t N) {
    launch_outer_product<float>(result, a, b, M, N);
}

void tensor_outer_product_float64(double* result, const double* a, const double* b, size_t M, size_t N) {
    launch_outer_product<double>(result, a, b, M, N);
}

// Matrix-vector multiplication operations
void tensor_matvec_float32(float* result, const float* A, const float* v, size_t M, size_t N) {
    const float alpha = 1.0f;
    const float beta  = 0.0f;
    hipblasHandle_t handle = cuda_utils::get_cublas_handle();
    hipblasStatus_t stat = hipblasSgemv(
        handle,
        HIPBLAS_OP_N,
        static_cast<int>(M), static_cast<int>(N),
        &alpha,
        A, static_cast<int>(M),
        v, 1,
        &beta,
        result, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        throw std::runtime_error("hipblasSgemv failed in tensor_matvec_float32");
    }
}

void tensor_matvec_float64(double* result, const double* A, const double* v, size_t M, size_t N) {
    const double alpha = 1.0;
    const double beta  = 0.0;
    hipblasHandle_t handle = cuda_utils::get_cublas_handle();
    hipblasStatus_t stat = hipblasDgemv(
        handle,
        HIPBLAS_OP_N,
        static_cast<int>(M), static_cast<int>(N),
        &alpha,
        A, static_cast<int>(M),
        v, 1,
        &beta,
        result, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        throw std::runtime_error("hipblasDgemv failed in tensor_matvec_float64");
    }
}

// Vector-matrix multiplication operations
void tensor_vecmat_float32(float* result, const float* v, const float* A, size_t M, size_t N) {
    const float alpha = 1.0f;
    const float beta  = 0.0f;
    hipblasHandle_t handle = cuda_utils::get_cublas_handle();
    // v^T (1×M) * A (M×N) = result (1×N)
    hipblasStatus_t stat = hipblasSgemv(
        handle,
        HIPBLAS_OP_T,
        static_cast<int>(M), static_cast<int>(N),
        &alpha,
        A, static_cast<int>(M),
        v, 1,
        &beta,
        result, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        throw std::runtime_error("hipblasSgemv (vecmat) failed in tensor_vecmat_float32");
    }
}

void tensor_vecmat_float64(double* result, const double* v, const double* A, size_t M, size_t N) {
    const double alpha = 1.0;
    const double beta  = 0.0;
    hipblasHandle_t handle = cuda_utils::get_cublas_handle();
    hipblasStatus_t stat = hipblasDgemv(
        handle,
        HIPBLAS_OP_T,
        static_cast<int>(M), static_cast<int>(N),
        &alpha,
        A, static_cast<int>(M),
        v, 1,
        &beta,
        result, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        throw std::runtime_error("hipblasDgemv (vecmat) failed in tensor_vecmat_float64");
    }
}

// Strided copy operations
void tensor_strided_copy_float32(float* dest, const float* src, const int* strides, const int* shape, int ndims, size_t total_elements) {
    std::vector<int> stride_vec(strides, strides + ndims);
    std::vector<int> shape_vec(shape, shape + ndims);
    launch_strided_copy<float>(dest, src, stride_vec, shape_vec, total_elements);
}

void tensor_strided_copy_float64(double* dest, const double* src, const int* strides, const int* shape, int ndims, size_t total_elements) {
    std::vector<int> stride_vec(strides, strides + ndims);
    std::vector<int> shape_vec(shape, shape + ndims);
    launch_strided_copy<double>(dest, src, stride_vec, shape_vec, total_elements);
}

// Strided operations - now fully implemented with strided kernels
void tensor_add_strided_float32(const cuda_utils::TensorDescriptor& out_desc,
                                const cuda_utils::TensorDescriptor& a_desc,
                                const cuda_utils::TensorDescriptor& b_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_binary_kernel<float, AddOp><<<grid_size, block_size>>>(
        out_desc, a_desc, b_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_add_strided_float64(const cuda_utils::TensorDescriptor& out_desc,
                                const cuda_utils::TensorDescriptor& a_desc,
                                const cuda_utils::TensorDescriptor& b_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_binary_kernel<double, AddOp><<<grid_size, block_size>>>(
        out_desc, a_desc, b_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_add_strided_float16(const cuda_utils::TensorDescriptor& out_desc,
                                const cuda_utils::TensorDescriptor& a_desc,
                                const cuda_utils::TensorDescriptor& b_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_binary_kernel<half, AddOp><<<grid_size, block_size>>>(
        out_desc, a_desc, b_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_add_strided_int8(const cuda_utils::TensorDescriptor& out_desc,
                            const cuda_utils::TensorDescriptor& a_desc,
                            const cuda_utils::TensorDescriptor& b_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_binary_kernel<int8_t, AddOp><<<grid_size, block_size>>>(
        out_desc, a_desc, b_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_exp_strided_float32(const cuda_utils::TensorDescriptor& out_desc, 
                                const cuda_utils::TensorDescriptor& in_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_unary_kernel<float, ExpOp><<<grid_size, block_size>>>(
        out_desc, in_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_exp_strided_float64(const cuda_utils::TensorDescriptor& out_desc, 
                                const cuda_utils::TensorDescriptor& in_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_unary_kernel<double, ExpOp><<<grid_size, block_size>>>(
        out_desc, in_desc, total_elements
    );
    hipDeviceSynchronize();
}

// Additional strided operations for more comprehensive support
void tensor_mul_strided_float32(const cuda_utils::TensorDescriptor& out_desc,
                                const cuda_utils::TensorDescriptor& a_desc,
                                const cuda_utils::TensorDescriptor& b_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_binary_kernel<float, MulOp><<<grid_size, block_size>>>(
        out_desc, a_desc, b_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_mul_strided_float64(const cuda_utils::TensorDescriptor& out_desc,
                                const cuda_utils::TensorDescriptor& a_desc,
                                const cuda_utils::TensorDescriptor& b_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_binary_kernel<double, MulOp><<<grid_size, block_size>>>(
        out_desc, a_desc, b_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_mul_strided_float16(const cuda_utils::TensorDescriptor& out_desc,
                                const cuda_utils::TensorDescriptor& a_desc,
                                const cuda_utils::TensorDescriptor& b_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_binary_kernel<half, MulOp><<<grid_size, block_size>>>(
        out_desc, a_desc, b_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_sub_strided_float32(const cuda_utils::TensorDescriptor& out_desc,
                                const cuda_utils::TensorDescriptor& a_desc,
                                const cuda_utils::TensorDescriptor& b_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_binary_kernel<float, SubOp><<<grid_size, block_size>>>(
        out_desc, a_desc, b_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_sub_strided_float64(const cuda_utils::TensorDescriptor& out_desc,
                                const cuda_utils::TensorDescriptor& a_desc,
                                const cuda_utils::TensorDescriptor& b_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_binary_kernel<double, SubOp><<<grid_size, block_size>>>(
        out_desc, a_desc, b_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_div_strided_float32(const cuda_utils::TensorDescriptor& out_desc,
                                const cuda_utils::TensorDescriptor& a_desc,
                                const cuda_utils::TensorDescriptor& b_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_binary_kernel<float, DivOp><<<grid_size, block_size>>>(
        out_desc, a_desc, b_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_div_strided_float64(const cuda_utils::TensorDescriptor& out_desc,
                                const cuda_utils::TensorDescriptor& a_desc,
                                const cuda_utils::TensorDescriptor& b_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_binary_kernel<double, DivOp><<<grid_size, block_size>>>(
        out_desc, a_desc, b_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_sub_strided_float16(const cuda_utils::TensorDescriptor& out_desc,
                                const cuda_utils::TensorDescriptor& a_desc,
                                const cuda_utils::TensorDescriptor& b_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_binary_kernel<half, SubOp><<<grid_size, block_size>>>(
        out_desc, a_desc, b_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_div_strided_float16(const cuda_utils::TensorDescriptor& out_desc,
                                const cuda_utils::TensorDescriptor& a_desc,
                                const cuda_utils::TensorDescriptor& b_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_binary_kernel<half, DivOp><<<grid_size, block_size>>>(
        out_desc, a_desc, b_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_log_strided_float32(const cuda_utils::TensorDescriptor& out_desc, 
                                const cuda_utils::TensorDescriptor& in_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_unary_kernel<float, LogOp><<<grid_size, block_size>>>(
        out_desc, in_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_log_strided_float64(const cuda_utils::TensorDescriptor& out_desc, 
                                const cuda_utils::TensorDescriptor& in_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_unary_kernel<double, LogOp><<<grid_size, block_size>>>(
        out_desc, in_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_sqrt_strided_float32(const cuda_utils::TensorDescriptor& out_desc, 
                                 const cuda_utils::TensorDescriptor& in_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_unary_kernel<float, SqrtOp><<<grid_size, block_size>>>(
        out_desc, in_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_sqrt_strided_float64(const cuda_utils::TensorDescriptor& out_desc, 
                                 const cuda_utils::TensorDescriptor& in_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_unary_kernel<double, SqrtOp><<<grid_size, block_size>>>(
        out_desc, in_desc, total_elements
    );
    hipDeviceSynchronize();
}

// Strided scalar operations - the key missing piece for view efficiency!
void tensor_scalar_add_strided_float32(const cuda_utils::TensorDescriptor& out_desc,
                                       const cuda_utils::TensorDescriptor& in_desc,
                                       float scalar) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_scalar_kernel<float, float, AddOp><<<grid_size, block_size>>>(
        out_desc, in_desc, scalar, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_scalar_add_strided_float64(const cuda_utils::TensorDescriptor& out_desc,
                                       const cuda_utils::TensorDescriptor& in_desc,
                                       double scalar) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_scalar_kernel<double, double, AddOp><<<grid_size, block_size>>>(
        out_desc, in_desc, scalar, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_scalar_mul_strided_float32(const cuda_utils::TensorDescriptor& out_desc,
                                       const cuda_utils::TensorDescriptor& in_desc,
                                       float scalar) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_scalar_kernel<float, float, MulOp><<<grid_size, block_size>>>(
        out_desc, in_desc, scalar, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_scalar_mul_strided_float64(const cuda_utils::TensorDescriptor& out_desc,
                                       const cuda_utils::TensorDescriptor& in_desc,
                                       double scalar) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_scalar_kernel<double, double, MulOp><<<grid_size, block_size>>>(
        out_desc, in_desc, scalar, total_elements
    );
    hipDeviceSynchronize();
}

// New strided math functions for Phase 3.1
void tensor_floor_strided_float32(const cuda_utils::TensorDescriptor& out_desc, 
                                  const cuda_utils::TensorDescriptor& in_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_unary_kernel<float, FloorOp><<<grid_size, block_size>>>(
        out_desc, in_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_floor_strided_float64(const cuda_utils::TensorDescriptor& out_desc, 
                                  const cuda_utils::TensorDescriptor& in_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_unary_kernel<double, FloorOp><<<grid_size, block_size>>>(
        out_desc, in_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_ceil_strided_float32(const cuda_utils::TensorDescriptor& out_desc, 
                                 const cuda_utils::TensorDescriptor& in_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_unary_kernel<float, CeilOp><<<grid_size, block_size>>>(
        out_desc, in_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_ceil_strided_float64(const cuda_utils::TensorDescriptor& out_desc, 
                                 const cuda_utils::TensorDescriptor& in_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_unary_kernel<double, CeilOp><<<grid_size, block_size>>>(
        out_desc, in_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_round_strided_float32(const cuda_utils::TensorDescriptor& out_desc, 
                                  const cuda_utils::TensorDescriptor& in_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_unary_kernel<float, RoundOp><<<grid_size, block_size>>>(
        out_desc, in_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_round_strided_float64(const cuda_utils::TensorDescriptor& out_desc, 
                                  const cuda_utils::TensorDescriptor& in_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_unary_kernel<double, RoundOp><<<grid_size, block_size>>>(
        out_desc, in_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_erf_strided_float32(const cuda_utils::TensorDescriptor& out_desc, 
                                const cuda_utils::TensorDescriptor& in_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_unary_kernel<float, ErfOp><<<grid_size, block_size>>>(
        out_desc, in_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_erf_strided_float64(const cuda_utils::TensorDescriptor& out_desc, 
                                const cuda_utils::TensorDescriptor& in_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_unary_kernel<double, ErfOp><<<grid_size, block_size>>>(
        out_desc, in_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_pow_scalar_strided_float32(const cuda_utils::TensorDescriptor& out_desc,
                                       const cuda_utils::TensorDescriptor& in_desc,
                                       float exponent) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_scalar_kernel<float, float, PowScalarOp><<<grid_size, block_size>>>(
        out_desc, in_desc, exponent, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_pow_scalar_strided_float64(const cuda_utils::TensorDescriptor& out_desc,
                                       const cuda_utils::TensorDescriptor& in_desc,
                                       double exponent) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_scalar_kernel<double, double, PowScalarOp><<<grid_size, block_size>>>(
        out_desc, in_desc, exponent, total_elements
    );
    hipDeviceSynchronize();
}

// New binary element-wise operations for Phase 3.2
void tensor_max_elemwise_float32(float* result, const float* a, const float* b, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_binary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, a, b, n, MaxOp());
    hipDeviceSynchronize();
}

void tensor_max_elemwise_float64(double* result, const double* a, const double* b, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_binary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, a, b, n, MaxOp());
    hipDeviceSynchronize();
}

void tensor_min_elemwise_float32(float* result, const float* a, const float* b, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_binary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, a, b, n, MinOp());
    hipDeviceSynchronize();
}

void tensor_min_elemwise_float64(double* result, const double* a, const double* b, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_binary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, a, b, n, MinOp());
    hipDeviceSynchronize();
}

void tensor_pow_elemwise_float32(float* result, const float* a, const float* b, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_binary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, a, b, n, PowOp());
    hipDeviceSynchronize();
}

void tensor_pow_elemwise_float64(double* result, const double* a, const double* b, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_binary_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, a, b, n, PowOp());
    hipDeviceSynchronize();
}

// New strided binary element-wise operations for Phase 3.2
void tensor_max_elemwise_strided_float32(const cuda_utils::TensorDescriptor& out_desc,
                                         const cuda_utils::TensorDescriptor& a_desc,
                                         const cuda_utils::TensorDescriptor& b_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_binary_kernel<float, MaxOp><<<grid_size, block_size>>>(
        out_desc, a_desc, b_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_max_elemwise_strided_float64(const cuda_utils::TensorDescriptor& out_desc,
                                         const cuda_utils::TensorDescriptor& a_desc,
                                         const cuda_utils::TensorDescriptor& b_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_binary_kernel<double, MaxOp><<<grid_size, block_size>>>(
        out_desc, a_desc, b_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_min_elemwise_strided_float32(const cuda_utils::TensorDescriptor& out_desc,
                                         const cuda_utils::TensorDescriptor& a_desc,
                                         const cuda_utils::TensorDescriptor& b_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_binary_kernel<float, MinOp><<<grid_size, block_size>>>(
        out_desc, a_desc, b_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_min_elemwise_strided_float64(const cuda_utils::TensorDescriptor& out_desc,
                                         const cuda_utils::TensorDescriptor& a_desc,
                                         const cuda_utils::TensorDescriptor& b_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_binary_kernel<double, MinOp><<<grid_size, block_size>>>(
        out_desc, a_desc, b_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_pow_elemwise_strided_float32(const cuda_utils::TensorDescriptor& out_desc,
                                         const cuda_utils::TensorDescriptor& a_desc,
                                         const cuda_utils::TensorDescriptor& b_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_binary_kernel<float, PowOp><<<grid_size, block_size>>>(
        out_desc, a_desc, b_desc, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_pow_elemwise_strided_float64(const cuda_utils::TensorDescriptor& out_desc,
                                         const cuda_utils::TensorDescriptor& a_desc,
                                         const cuda_utils::TensorDescriptor& b_desc) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_binary_kernel<double, PowOp><<<grid_size, block_size>>>(
        out_desc, a_desc, b_desc, total_elements
    );
    hipDeviceSynchronize();
}

// Product reductions

// ... existing code ...

// New scalar operations for Phase 3.2
void tensor_scalar_sub_float32(float* result, const float* input, float scalar, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_scalar_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, -scalar, n, AddOp());
    hipDeviceSynchronize();
}

void tensor_scalar_sub_float64(double* result, const double* input, double scalar, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_scalar_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, -scalar, n, AddOp());
    hipDeviceSynchronize();
}

void tensor_scalar_div_float32(float* result, const float* input, float scalar, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_scalar_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, 1.0f / scalar, n, MulOp());
    hipDeviceSynchronize();
}

void tensor_scalar_div_float64(double* result, const double* input, double scalar, size_t n) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_scalar_kernel<<<blocksPerGrid, threadsPerBlock>>>(result, input, 1.0 / scalar, n, MulOp());
    hipDeviceSynchronize();
}

// New strided scalar operations for Phase 3.2
void tensor_scalar_sub_strided_float32(const cuda_utils::TensorDescriptor& out_desc,
                                       const cuda_utils::TensorDescriptor& in_desc,
                                       float scalar) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_scalar_kernel<float, float, AddOp><<<grid_size, block_size>>>(
        out_desc, in_desc, -scalar, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_scalar_sub_strided_float64(const cuda_utils::TensorDescriptor& out_desc,
                                       const cuda_utils::TensorDescriptor& in_desc,
                                       double scalar) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_scalar_kernel<double, double, AddOp><<<grid_size, block_size>>>(
        out_desc, in_desc, -scalar, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_scalar_div_strided_float32(const cuda_utils::TensorDescriptor& out_desc,
                                       const cuda_utils::TensorDescriptor& in_desc,
                                       float scalar) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_scalar_kernel<float, float, MulOp><<<grid_size, block_size>>>(
        out_desc, in_desc, 1.0f / scalar, total_elements
    );
    hipDeviceSynchronize();
}

void tensor_scalar_div_strided_float64(const cuda_utils::TensorDescriptor& out_desc,
                                       const cuda_utils::TensorDescriptor& in_desc,
                                       double scalar) {
    size_t total_elements = out_desc.total_size;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    
    strided_scalar_kernel<double, double, MulOp><<<grid_size, block_size>>>(
        out_desc, in_desc, 1.0 / scalar, total_elements
    );
    hipDeviceSynchronize();
}

// New scalar operations for Phase 3.2

// ... existing code ...

} // extern "C"